#include "hip/hip_runtime.h"
#include <array>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "noarr_funcs.hpp"

using namespace noarr;

// same body, two data layouts:
template<typename AS>
__global__ void kernel(float *data, AS as) {
    auto index = as % fixs<'y', 'x'>(blockIdx.x, threadIdx.x);
    *(float*)((char*)data + (index % offset())) = blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void kernel_handmade(float *data, size_t size) {
    data[blockIdx.x * size + threadIdx.x] = blockIdx.x * blockDim.x + threadIdx.x;
}

int main() {
    float *data;
    std::cout << "hi" << std::endl;

    std::array<float, 40000> local;
    

    hipMalloc(&data, sizeof(local));

    // kernel<<<20000, 20>>>(data, (scalar<float> ^ vector<'y'> ^ array<'x', 20000>) % resize<'y'>(20));
    const auto av = array<'y', 2000, vector<'x', scalar<float>>>{};
    volatile std::size_t s = 20;
    const auto avr = av % resize<'x'>(s);
    kernel<<<2000, 20>>>(data, avr);
    //kernel_handmade<<<2000, 20>>>(data, 20);
    
    hipMemcpy(local.data(), data, sizeof(local), hipMemcpyDeviceToHost);
    
    size_t i = 0;
    for (auto f : local) {
        std::cout << f << ((i++ % 25 == 24) ? '\n' : ' ');
    }

    std::cout.flush();

    hipFree(data);
}
