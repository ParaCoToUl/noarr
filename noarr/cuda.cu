#include "hip/hip_runtime.h"
#include <iostream>

#include "noarr_funcs.hpp"

using namespace noarr;

// same body, two data layouts:
template<typename AS>
__global__ void kernel(float *data, AS as) {
    auto index = as % fixs<'y', 'x'>(blockIdx.x, threadIdx.x);
    *(float*)((char*)data + (index % offset())) = blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void kernel_handmade(float *data, size_t size) {
    data[blockIdx.x * size + threadIdx.x] = blockIdx.x * blockDim.x + threadIdx.x;
}

int main() {
    float *data;

    std::array<float, 400000> local;

    hipMalloc(&data, sizeof(local));

    // kernel<<<20000, 20>>>(data, (scalar<float> ^ vector<'y'> ^ array<'x', 20000>) % resize<'y'>(20));
    const auto av = array<'y', 20000, vector<'x', scalar<float>>>{};
    volatile std::size_t s = 20;
    const auto avr = av % resize<'x'>(s);
    kernel<<<20000, 20>>>(data, avr);
    //kernel_handmade<<<20000, 20>>>(data, 20);
    
    hipMemcpy(local.data(), data, sizeof(local), hipMemcpyDeviceToHost);
    
    size_t i = 0;
    for (auto f : local) {
        std::cout << f << ((i++ % 25 == 24) ? '\n' : ' ');
    }

    std::cout.flush();

    hipFree(data);
}
